#define INFINITY (__int_as_float(0x7f800000))
#define NAN (__int_as_float(0x7fffffff))

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cuda_pipeline.h>
#define N_PAD 132

struct __align__(8) half4 { half x, y, z, w; };
__device__ half4 make_half4(half x, half y, half z, half w) { half4 r={x, y, z, w}; return r; }

struct __align__(16) half8 { half x, y, z, w, a, b, c, d; };
__device__ half8 make_half8(half x, half y, half z, half w, half a, half b, half c, half d) { half8 r={x, y, z, w, a, b, c, d}; return r; }

__device__ void __ldmatrix_a_elems(half8 *regs, half *smem) {
    uint32_t reg0, reg1, reg2, reg3;
    asm volatile(
        "ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];"
        : "=r"(reg0), "=r"(reg1), "=r"(reg2), "=r"(reg3)
        : "l"(__cvta_generic_to_shared(smem))
    );
    uint32_t *addr = reinterpret_cast<uint32_t*>(regs);
    addr[0] = reg0;
    addr[1] = reg1;
    addr[2] = reg2;
    addr[3] = reg3;
}

__device__ void __ldmatrix_b_elems(half4 *regs_lo, half4 *regs_hi, half *smem) {
    uint32_t reg0, reg1, reg2, reg3;
    asm volatile(
        "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16 {%0, %1, %2, %3}, [%4];"
        : "=r"(reg0), "=r"(reg1), "=r"(reg2), "=r"(reg3)
        : "l"(__cvta_generic_to_shared(smem))
    );
    uint32_t *addr_lo = reinterpret_cast<uint32_t*>(regs_lo);
    uint32_t *addr_hi = reinterpret_cast<uint32_t*>(regs_hi);
    addr_lo[0] = reg0;
    addr_lo[1] = reg1;
    addr_hi[0] = reg2;
    addr_hi[1] = reg3;
}

__device__ float4 __WMMA_8_16_16_half_float(half8 a, half4 b, float4 c) {
    int *a_pk = (int *) (&a), *b_pk = (int *) (&b);
    asm( "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 { %0, %1, %2, %3 }, { %4, %5, %6, %7 }, { %8, %9 }, { %0, %1, %2, %3 };"
        : "+f"(c.x), "+f"(c.y), "+f"(c.z), "+f"(c.w) : "r"(a_pk[0]), "r"(a_pk[1]), "r"(a_pk[2]),  "r"(a_pk[3]), "r"(b_pk[0]), "r"(b_pk[1]) );
    return c;
}

extern "C" __global__ void __launch_bounds__(128) wmma_example(float* data0, const half* data1, const half* data2, int N, int K) {
    int grid_m = blockIdx.x;        /* M//64 */
    int grid_n = blockIdx.y;        /* N//128 */
    int threads = threadIdx.x;      /* 128 */
    int wg_m = (threads/64);        // 0 or 1 for 1st and 3rd blocks of b_m=16xb_k=16 vs 2nd and 4th blocks
    int wg_n = (threads/32)%2;      // 0 or 1 for 1st, 3rd, 5th, 7th blocks of b_n=16xb_k=16 vs 2nd, 4th, 6th, 8th blocks - differs from triton
    int wg_threads = threads%32;
    int num_k_blocks = K / 64;

    // load indexes
    size_t global_a_off = ((grid_m * 64) * K) + ((threads %  8) * 8) + ((threads /  8) * K);
    size_t global_b_off = (grid_n * 128)      + ((threads % 16) * 8) + ((threads / 16) * N);

    // non-swizzled - should work slowly with bank conflicts
    size_t store_smem_a_off = ((threads %  8) * 8) + ((threads /  8) *  64);
    size_t store_smem_b_off = ((threads % 16) * 8) + ((threads / 16) * 128);

    // ldmatrix indices
    // threads 0-7 are row starts for A, 8-15 for B, 16-23 for C, 24-31 for D
    // [ A | C ]
    // [ - + - ]
    // [ B | D ]

    // unswizzled ldmatrix
    size_t load_smem_a_0_k_0 = (wg_m * 16 * 64) + ((wg_threads % 8) *  64) + (((wg_threads / 8) % 2) *  64 * 8) + ((wg_threads / 16) * 8);
    size_t load_smem_a_1_k_0 = load_smem_a_0_k_0 + (32*64);
    size_t load_smem_b_0_k_0 = (wg_n * 16)      + ((wg_threads % 8) * 128) + (((wg_threads / 8) % 2) * 128 * 8) + ((wg_threads / 16) * 8);
    size_t load_smem_b_1_k_0 = load_smem_b_0_k_0 + 32;
    size_t load_smem_b_2_k_0 = load_smem_b_0_k_0 + 64;
    size_t load_smem_b_3_k_0 = load_smem_b_0_k_0 + 96;

    size_t load_smem_a_0_k_1 = load_smem_a_0_k_0 + 16;
    size_t load_smem_a_1_k_1 = load_smem_a_1_k_0 + 16;
    size_t load_smem_b_0_k_1 = load_smem_b_0_k_0 + (16 * 128);
    size_t load_smem_b_1_k_1 = load_smem_b_1_k_0 + (16 * 128);
    size_t load_smem_b_2_k_1 = load_smem_b_2_k_0 + (16 * 128);
    size_t load_smem_b_3_k_1 = load_smem_b_3_k_0 + (16 * 128);

    size_t load_smem_a_0_k_2 = load_smem_a_0_k_0 + 32;
    size_t load_smem_a_1_k_2 = load_smem_a_1_k_0 + 32;
    size_t load_smem_b_0_k_2 = load_smem_b_0_k_0 + (32 * 128);
    size_t load_smem_b_1_k_2 = load_smem_b_1_k_0 + (32 * 128);
    size_t load_smem_b_2_k_2 = load_smem_b_2_k_0 + (32 * 128);
    size_t load_smem_b_3_k_2 = load_smem_b_3_k_0 + (32 * 128);

    size_t load_smem_a_0_k_3 = load_smem_a_0_k_0 + 48;
    size_t load_smem_a_1_k_3 = load_smem_a_1_k_0 + 48;
    size_t load_smem_b_0_k_3 = load_smem_b_0_k_0 + (48 * 128);
    size_t load_smem_b_1_k_3 = load_smem_b_1_k_0 + (48 * 128);
    size_t load_smem_b_2_k_3 = load_smem_b_2_k_0 + (48 * 128);
    size_t load_smem_b_3_k_3 = load_smem_b_3_k_0 + (48 * 128);

    // create shared mem (A 8192 bytes, B 16384 bytes)
    __shared__ alignas(16) char smem[24576];

    // create accs (16 WMMAs and 4 output elements each) and zero
    float4 acc_frag_0_0 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_1 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_2 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_3 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_4 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_5 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_6 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_0_7 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_0 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_1 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_2 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_3 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_4 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_5 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_6 = make_float4(0.0f,0.0f,0.0f,0.0f);
    float4 acc_frag_1_7 = make_float4(0.0f,0.0f,0.0f,0.0f);

    // create registers for block A elements (2)
    half8 a_frag_0;
    half8 a_frag_1;

    // create register for block B elements (8)
    half4 b_frag_0;
    half4 b_frag_1;
    half4 b_frag_2;
    half4 b_frag_3;
    half4 b_frag_4;
    half4 b_frag_5;
    half4 b_frag_6;
    half4 b_frag_7;

    half *smem_a = (half *)(smem);
    half *smem_b = (half *)(smem + 8192);

    // https://developer.nvidia.com/blog/controlling-data-movement-to-boost-performance-on-ampere-architecture/
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#asynchronous-data-copies

    // start first pre-fetch load A
    __pipeline_memcpy_async(&smem_a[store_smem_a_off +  (    0)], &data1[global_a_off + (   0)], 16);
    __pipeline_memcpy_async(&smem_a[store_smem_a_off +  (16*64)], &data1[global_a_off + (16*K)], 16);
    __pipeline_memcpy_async(&smem_a[store_smem_a_off +  (32*64)], &data1[global_a_off + (32*K)], 16);
    __pipeline_memcpy_async(&smem_a[store_smem_a_off +  (48*64)], &data1[global_a_off + (48*K)], 16);

    // start first pre-fetch load B
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (     0)], &data2[global_b_off + (   0)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + ( 8*128)], &data2[global_b_off + ( 8*N)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (16*128)], &data2[global_b_off + (16*N)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (24*128)], &data2[global_b_off + (24*N)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (32*128)], &data2[global_b_off + (32*N)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (40*128)], &data2[global_b_off + (40*N)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (48*128)], &data2[global_b_off + (48*N)], 16);
    __pipeline_memcpy_async(&smem_b[store_smem_b_off + (56*128)], &data2[global_b_off + (56*N)], 16);
    __pipeline_commit();

    global_a_off += 64;
    global_b_off += 64 * N;
    __syncthreads();

    for (int block_k = 0; block_k < num_k_blocks; block_k++) {
        // wait on needed prefetch value
        __pipeline_wait_prior(0);
        __syncthreads();

        // BLOCK_K==4: unroll 4 iterations of ldmatrix/wmma
        half *smem_a_curr = smem_a;
        half *smem_b_curr = smem_b;

        // first load 16 K elements and 16 WMMAs: BLOCK_M==2 * BLOCK_N==8
        __ldmatrix_a_elems(&a_frag_0,            &smem_a_curr[load_smem_a_0_k_0]);
        __ldmatrix_a_elems(&a_frag_1,            &smem_a_curr[load_smem_a_1_k_0]);
        __ldmatrix_b_elems(&b_frag_0, &b_frag_1, &smem_b_curr[load_smem_b_0_k_0]);
        __ldmatrix_b_elems(&b_frag_2, &b_frag_3, &smem_b_curr[load_smem_b_1_k_0]);
        __ldmatrix_b_elems(&b_frag_4, &b_frag_5, &smem_b_curr[load_smem_b_2_k_0]);
        __ldmatrix_b_elems(&b_frag_6, &b_frag_7, &smem_b_curr[load_smem_b_3_k_0]);
        acc_frag_0_0 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_0, acc_frag_0_0);
        acc_frag_0_1 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_1, acc_frag_0_1);
        acc_frag_0_2 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_2, acc_frag_0_2);
        acc_frag_0_3 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_3, acc_frag_0_3);
        acc_frag_0_4 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_4, acc_frag_0_4);
        acc_frag_0_5 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_5, acc_frag_0_5);
        acc_frag_0_6 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_6, acc_frag_0_6);
        acc_frag_0_7 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_7, acc_frag_0_7);
        acc_frag_1_0 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_0, acc_frag_1_0);
        acc_frag_1_1 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_1, acc_frag_1_1);
        acc_frag_1_2 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_2, acc_frag_1_2);
        acc_frag_1_3 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_3, acc_frag_1_3);
        acc_frag_1_4 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_4, acc_frag_1_4);
        acc_frag_1_5 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_5, acc_frag_1_5);
        acc_frag_1_6 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_6, acc_frag_1_6);
        acc_frag_1_7 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_7, acc_frag_1_7);

        // next 16 K elements
        __ldmatrix_a_elems(&a_frag_0,            &smem_a_curr[load_smem_a_0_k_1]);
        __ldmatrix_a_elems(&a_frag_1,            &smem_a_curr[load_smem_a_1_k_1]);
        __ldmatrix_b_elems(&b_frag_0, &b_frag_1, &smem_b_curr[load_smem_b_0_k_1]);
        __ldmatrix_b_elems(&b_frag_2, &b_frag_3, &smem_b_curr[load_smem_b_1_k_1]);
        __ldmatrix_b_elems(&b_frag_4, &b_frag_5, &smem_b_curr[load_smem_b_2_k_1]);
        __ldmatrix_b_elems(&b_frag_6, &b_frag_7, &smem_b_curr[load_smem_b_3_k_1]);
        acc_frag_0_0 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_0, acc_frag_0_0);
        acc_frag_0_1 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_1, acc_frag_0_1);
        acc_frag_0_2 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_2, acc_frag_0_2);
        acc_frag_0_3 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_3, acc_frag_0_3);
        acc_frag_0_4 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_4, acc_frag_0_4);
        acc_frag_0_5 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_5, acc_frag_0_5);
        acc_frag_0_6 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_6, acc_frag_0_6);
        acc_frag_0_7 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_7, acc_frag_0_7);
        acc_frag_1_0 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_0, acc_frag_1_0);
        acc_frag_1_1 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_1, acc_frag_1_1);
        acc_frag_1_2 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_2, acc_frag_1_2);
        acc_frag_1_3 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_3, acc_frag_1_3);
        acc_frag_1_4 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_4, acc_frag_1_4);
        acc_frag_1_5 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_5, acc_frag_1_5);
        acc_frag_1_6 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_6, acc_frag_1_6);
        acc_frag_1_7 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_7, acc_frag_1_7);

        // next 16 K elements
        __ldmatrix_a_elems(&a_frag_0,            &smem_a_curr[load_smem_a_0_k_2]);
        __ldmatrix_a_elems(&a_frag_1,            &smem_a_curr[load_smem_a_1_k_2]);
        __ldmatrix_b_elems(&b_frag_0, &b_frag_1, &smem_b_curr[load_smem_b_0_k_2]);
        __ldmatrix_b_elems(&b_frag_2, &b_frag_3, &smem_b_curr[load_smem_b_1_k_2]);
        __ldmatrix_b_elems(&b_frag_4, &b_frag_5, &smem_b_curr[load_smem_b_2_k_2]);
        __ldmatrix_b_elems(&b_frag_6, &b_frag_7, &smem_b_curr[load_smem_b_3_k_2]);
        acc_frag_0_0 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_0, acc_frag_0_0);
        acc_frag_0_1 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_1, acc_frag_0_1);
        acc_frag_0_2 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_2, acc_frag_0_2);
        acc_frag_0_3 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_3, acc_frag_0_3);
        acc_frag_0_4 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_4, acc_frag_0_4);
        acc_frag_0_5 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_5, acc_frag_0_5);
        acc_frag_0_6 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_6, acc_frag_0_6);
        acc_frag_0_7 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_7, acc_frag_0_7);
        acc_frag_1_0 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_0, acc_frag_1_0);
        acc_frag_1_1 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_1, acc_frag_1_1);
        acc_frag_1_2 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_2, acc_frag_1_2);
        acc_frag_1_3 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_3, acc_frag_1_3);
        acc_frag_1_4 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_4, acc_frag_1_4);
        acc_frag_1_5 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_5, acc_frag_1_5);
        acc_frag_1_6 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_6, acc_frag_1_6);
        acc_frag_1_7 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_7, acc_frag_1_7);

        // last 16 K elements
        __ldmatrix_a_elems(&a_frag_0,            &smem_a_curr[load_smem_a_0_k_3]);
        __ldmatrix_a_elems(&a_frag_1,            &smem_a_curr[load_smem_a_1_k_3]);
        __ldmatrix_b_elems(&b_frag_0, &b_frag_1, &smem_b_curr[load_smem_b_0_k_3]);
        __ldmatrix_b_elems(&b_frag_2, &b_frag_3, &smem_b_curr[load_smem_b_1_k_3]);
        __ldmatrix_b_elems(&b_frag_4, &b_frag_5, &smem_b_curr[load_smem_b_2_k_3]);
        __ldmatrix_b_elems(&b_frag_6, &b_frag_7, &smem_b_curr[load_smem_b_3_k_3]);
        acc_frag_0_0 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_0, acc_frag_0_0);
        acc_frag_0_1 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_1, acc_frag_0_1);
        acc_frag_0_2 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_2, acc_frag_0_2);
        acc_frag_0_3 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_3, acc_frag_0_3);
        acc_frag_0_4 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_4, acc_frag_0_4);
        acc_frag_0_5 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_5, acc_frag_0_5);
        acc_frag_0_6 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_6, acc_frag_0_6);
        acc_frag_0_7 = __WMMA_8_16_16_half_float(a_frag_0, b_frag_7, acc_frag_0_7);
        acc_frag_1_0 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_0, acc_frag_1_0);
        acc_frag_1_1 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_1, acc_frag_1_1);
        acc_frag_1_2 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_2, acc_frag_1_2);
        acc_frag_1_3 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_3, acc_frag_1_3);
        acc_frag_1_4 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_4, acc_frag_1_4);
        acc_frag_1_5 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_5, acc_frag_1_5);
        acc_frag_1_6 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_6, acc_frag_1_6);
        acc_frag_1_7 = __WMMA_8_16_16_half_float(a_frag_1, b_frag_7, acc_frag_1_7);

        // prefetch next iteration if needed
        __syncthreads();
        if (block_k < (num_k_blocks-1)) {
            __pipeline_memcpy_async(&smem_a_curr[store_smem_a_off +  (    0)], &data1[global_a_off + (   0)], 16);
            __pipeline_memcpy_async(&smem_a_curr[store_smem_a_off +  (16*64)], &data1[global_a_off + (16*K)], 16);
            __pipeline_memcpy_async(&smem_a_curr[store_smem_a_off +  (32*64)], &data1[global_a_off + (32*K)], 16);
            __pipeline_memcpy_async(&smem_a_curr[store_smem_a_off +  (48*64)], &data1[global_a_off + (48*K)], 16);

            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (     0)], &data2[global_b_off + (   0)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + ( 8*128)], &data2[global_b_off + ( 8*N)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (16*128)], &data2[global_b_off + (16*N)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (24*128)], &data2[global_b_off + (24*N)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (32*128)], &data2[global_b_off + (32*N)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (40*128)], &data2[global_b_off + (40*N)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (48*128)], &data2[global_b_off + (48*N)], 16);
            __pipeline_memcpy_async(&smem_b_curr[store_smem_b_off + (56*128)], &data2[global_b_off + (56*N)], 16);

            global_a_off += 64;
            global_b_off += 64 * N;
        }
        __pipeline_commit();
    }

    // write accumulators to output
    __pipeline_wait_prior(0);
    __syncthreads();

    // slower way: write floats one by one to data0
    size_t wg_c_off     = ((grid_m * 64) * N) + (grid_n * 128) + (wg_m * 16 * N) + (wg_n * 16);
    size_t thread_c_off = ((wg_threads % 4) * 2) + (((wg_threads / 4) % 8) * N);
    data0[wg_c_off + thread_c_off           + 0 + ( 0*8)] = acc_frag_0_0.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 0*8)] = acc_frag_0_0.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 0*8)] = acc_frag_0_0.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 0*8)] = acc_frag_0_0.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 1*8)] = acc_frag_0_1.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 1*8)] = acc_frag_0_1.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 1*8)] = acc_frag_0_1.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 1*8)] = acc_frag_0_1.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 4*8)] = acc_frag_0_2.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 4*8)] = acc_frag_0_2.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 4*8)] = acc_frag_0_2.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 4*8)] = acc_frag_0_2.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 5*8)] = acc_frag_0_3.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 5*8)] = acc_frag_0_3.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 5*8)] = acc_frag_0_3.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 5*8)] = acc_frag_0_3.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 8*8)] = acc_frag_0_4.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 8*8)] = acc_frag_0_4.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 8*8)] = acc_frag_0_4.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 8*8)] = acc_frag_0_4.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 9*8)] = acc_frag_0_5.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 9*8)] = acc_frag_0_5.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 9*8)] = acc_frag_0_5.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 9*8)] = acc_frag_0_5.w;
    data0[wg_c_off + thread_c_off           + 0 + (12*8)] = acc_frag_0_6.x;
    data0[wg_c_off + thread_c_off           + 1 + (12*8)] = acc_frag_0_6.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + (12*8)] = acc_frag_0_6.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + (12*8)] = acc_frag_0_6.w;
    data0[wg_c_off + thread_c_off           + 0 + (13*8)] = acc_frag_0_7.x;
    data0[wg_c_off + thread_c_off           + 1 + (13*8)] = acc_frag_0_7.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + (13*8)] = acc_frag_0_7.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + (13*8)] = acc_frag_0_7.w;
    wg_c_off += 32*N;
    data0[wg_c_off + thread_c_off           + 0 + ( 0*8)] = acc_frag_1_0.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 0*8)] = acc_frag_1_0.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 0*8)] = acc_frag_1_0.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 0*8)] = acc_frag_1_0.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 1*8)] = acc_frag_1_1.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 1*8)] = acc_frag_1_1.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 1*8)] = acc_frag_1_1.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 1*8)] = acc_frag_1_1.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 4*8)] = acc_frag_1_2.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 4*8)] = acc_frag_1_2.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 4*8)] = acc_frag_1_2.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 4*8)] = acc_frag_1_2.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 5*8)] = acc_frag_1_3.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 5*8)] = acc_frag_1_3.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 5*8)] = acc_frag_1_3.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 5*8)] = acc_frag_1_3.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 8*8)] = acc_frag_1_4.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 8*8)] = acc_frag_1_4.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 8*8)] = acc_frag_1_4.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 8*8)] = acc_frag_1_4.w;
    data0[wg_c_off + thread_c_off           + 0 + ( 9*8)] = acc_frag_1_5.x;
    data0[wg_c_off + thread_c_off           + 1 + ( 9*8)] = acc_frag_1_5.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + ( 9*8)] = acc_frag_1_5.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + ( 9*8)] = acc_frag_1_5.w;
    data0[wg_c_off + thread_c_off           + 0 + (12*8)] = acc_frag_1_6.x;
    data0[wg_c_off + thread_c_off           + 1 + (12*8)] = acc_frag_1_6.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + (12*8)] = acc_frag_1_6.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + (12*8)] = acc_frag_1_6.w;
    data0[wg_c_off + thread_c_off           + 0 + (13*8)] = acc_frag_1_7.x;
    data0[wg_c_off + thread_c_off           + 1 + (13*8)] = acc_frag_1_7.y;
    data0[wg_c_off + thread_c_off + (8 * N) + 0 + (13*8)] = acc_frag_1_7.z;
    data0[wg_c_off + thread_c_off + (8 * N) + 1 + (13*8)] = acc_frag_1_7.w;
}
